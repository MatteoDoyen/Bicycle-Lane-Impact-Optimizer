#include "hip/hip_runtime.h"
#include "../header/dijkstra.h"
#include "../header/dijkstra_cuda.cuh"
#include <stdbool.h>
#include <stdlib.h>

void allocate_graph_on_gpu(vertex_t **graph_gpu, vertex_t **graph_cpu, uint32_t num_vertices)
{

    hipMalloc(graph_gpu, num_vertices * sizeof(vertex_t *));
    hipMemcpy(graph_gpu, graph_cpu, num_vertices * sizeof(vertex_t *), hipMemcpyHostToDevice);
    printf("allocate_graph %d\n",num_vertices);
    // Allocate memory for each vertex and edge on the GPU
    for (uint32_t i = 0; i < num_vertices; i++)
    {

        printf("boucle %d\n", graph_cpu[i]->nb_edges_in);
        if (graph_cpu[i]->nb_edges_in > 0)
        {
            hipMalloc(graph_gpu[i]->in_edges, graph_cpu[i]->nb_edges_in * sizeof(edge_t *));
            hipMemcpy(graph_gpu[i]->in_edges, graph_cpu[i]->in_edges, graph_cpu[i]->nb_edges_in * sizeof(edge_t *), hipMemcpyHostToDevice);
        }

        printf("midle\n");
        if (graph_cpu[i]->nb_edges_out > 0)
        {
            hipMalloc(graph_gpu[i]->out_edges, graph_cpu[i]->nb_edges_out * sizeof(edge_t *));
            hipMemcpy(graph_gpu[i]->out_edges, graph_cpu[i]->out_edges, graph_cpu[i]->nb_edges_out * sizeof(edge_t *), hipMemcpyHostToDevice);
        }
    }
}

void free_path_on_gpu(path_t *d_path)
{
    // Free memory on GPU
    hipFree(d_path->chemin);
    hipFree(d_path->dijkstra_sp);
    hipFree(d_path->visibilite);
}

void allocate_path_on_gpu(path_t **d_path, path_t *h_path)
{
    // Copy simple data types
    hipMemcpy(&((*d_path)->origin), &(h_path->origin), sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->destination), &(h_path->destination), sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->profil), &(h_path->profil), sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->distance), &(h_path->distance), sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->danger), &(h_path->danger), sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->nb_dijkstra_sp), &(h_path->nb_dijkstra_sp), sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->nb_chemin), &(h_path->nb_chemin), sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->nb_visibilite), &(h_path->nb_visibilite), sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->dijkstra_dist), &(h_path->dijkstra_dist), sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->cps_dijkstra_dist), &(h_path->cps_dijkstra_dist), sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&((*d_path)->cps_dijkstra_danger), &(h_path->cps_dijkstra_danger), sizeof(double), hipMemcpyHostToDevice);

    // Allocate memory for arrays on GPU
    hipMalloc((void **)&((*d_path)->chemin), h_path->nb_chemin * sizeof(uint32_t));
    hipMalloc((void **)&((*d_path)->dijkstra_sp), h_path->nb_dijkstra_sp * sizeof(uint32_t));
    hipMalloc((void **)&((*d_path)->visibilite), h_path->nb_visibilite * sizeof(uint32_t));
}

double dijkstra_forward_cuda(vertex_t **graph, int nb_vertices, double **dist_array_ref, int *parents, path_t *path)
{
    printf("la la\n");
    double *dist_array = *dist_array_ref;
    bool marked_vertex[nb_vertices];
    unsigned int origin = path->destination;
    unsigned int destination = path->origin;

    vertex_t **gpu_graph;
    double *gpu_dist_array;
    int *gpu_parents;
    path_t *gpu_path;
    bool *gpu_marked_vertex;
    int *gpu_nb_vertices;

    int graph_size = (sizeof(vertex_t *) * nb_vertices) + (sizeof(vertex_t) * nb_vertices);
    int dist_array_size = nb_vertices * sizeof(double);
    int marked_vertex_size = nb_vertices * sizeof(bool);
    int parents_size = nb_vertices * sizeof(int);
    int path_size = nb_vertices * sizeof(path_t);

    // Min:  One thread checks for closest vertex. Ideally there would be multiple threads working in
    //  parallel, but due to compiler issues with prallelized-reduction functions this is being used as a backup.
    dim3 gridMin(1, 1, 1);
    dim3 blockMin(1, 1, 1);

    // Relax: Each thread is responsible for relaxing from a shared, given vertex
    //   to one other vertex determined by the ID of the thread. Since each thread handles
    //   a different vertex, there's no RaW or WaR data hazards; all that's needed is a
    //   __syncthreads(); call at the end to ensure all either update or do nothing.
    dim3 gridRelax(nb_vertices / THREADS_BLOCK, 1, 1);
    dim3 blockRelax(THREADS_BLOCK, 1, 1);

    // for closest vertex
    int *closest_vertex = (int *)malloc(sizeof(int));
    int *gpu_closest_vertex;
    closest_vertex[0] = -1;
    hipMalloc((void **)&gpu_closest_vertex, (sizeof(int)));
    hipMemcpy(gpu_closest_vertex, closest_vertex, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&gpu_nb_vertices, (sizeof(int)));
    hipMemcpy(&gpu_nb_vertices[0], &nb_vertices, sizeof(int), hipMemcpyHostToDevice);
    printf("2\n");
    allocate_path_on_gpu(&gpu_path, path);
    allocate_graph_on_gpu(gpu_graph, graph, nb_vertices);
    printf("3\n");
    hipMalloc((void **)&gpu_dist_array, dist_array_size);
    hipMalloc((void **)&gpu_marked_vertex, marked_vertex_size);
    hipMalloc((void **)&gpu_parents, parents_size);
    hipMalloc((void **)&gpu_path, parents_size);
    printf("4\n");
    hipEvent_t exec_start, exec_stop; // timer for execution only
    float elapsed_exec;                // elapsed time
    hipEventCreate(&exec_start);
    hipEventCreate(&exec_stop);

    // Initialize distances, set all vertices as not yet included in the shortest path tree
    for (int i = 0; i < nb_vertices; i++)
    {
        dist_array[i] = DBL_MAX;
        marked_vertex[i] = false;
    }

    // Distance from source to itself is always 0
    dist_array[origin] = 0;
    parents[origin] = -1;

    // start distance is zero; ensures it will be first pulled out
    // gpu source        cpu source      memory size     HtD or DtH
    // hipMemcpy(gpu_graph, graph, graph_size, hipMemcpyHostToDevice));

    hipMemcpy(gpu_dist_array, dist_array, dist_array_size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_marked_vertex, marked_vertex, marked_vertex_size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_parents, parents, parents_size, hipMemcpyHostToDevice);

    printf("avant\n");
    hipEventRecord(exec_start);
    for (int i = 0; i < nb_vertices; i++)
    {
        min_distance_cuda<<<gridMin, blockMin>>>(gpu_dist_array, gpu_marked_vertex, gpu_closest_vertex, gpu_nb_vertices);              // find min
        cudaRelax<<<gridRelax, blockRelax>>>(gpu_graph, gpu_dist_array, gpu_parents, gpu_marked_vertex, gpu_closest_vertex, gpu_path); // relax
        // cudaRelax(vertex_t * *graph, double *dist_array, int *parents, bool *marked_vertex, int *global_min_index, path_t *path)
    }
    hipEventRecord(exec_stop);

    // // save data in PN, ND matrices
    // hipMemcpy(node_dist, gpu_node_dist, data_array, hipMemcpyDeviceToHost));
    // hipMemcpy(parent_node, gpu_parent_node, int_array, hipMemcpyDeviceToHost));
    // hipMemcpy(visited_node, gpu_visited_node, int_array, hipMemcpyDeviceToHost));

    // for (i = 0; i < VERTICES; i++) {                //record resulting parent array and node distance
    //     pn_matrix[version*VERTICES + i] = parent_node[i];
    //     dist_matrix[version*VERTICES + i] = node_dist[i];
    // }

    // free memory
    hipFree(gpu_graph);
    hipFree(gpu_dist_array);
    hipFree(gpu_marked_vertex);
    hipFree(gpu_parents);
    free_path_on_gpu(gpu_path);

    // return dist_array[destination];
    return 0.0;
}

__global__ void min_distance_cuda(double *dist, bool *marked_vertex, int *global_min_index, int *nb_vertices)
{
    double min = DBL_MAX;
    int min_index = -1;
    int i;

    for (i = 0; i < nb_vertices[0]; i++)
    {
        if ((dist[i] < min) && (marked_vertex[i] != 1))
        {
            min = dist[i];
            min_index = i;
        }
    }

    global_min_index[0] = min_index;
    marked_vertex[min_index] = 1;
}

__global__ void cudaRelax(vertex_t **graph, double *dist_array, int *parents, bool *marked_vertex, int *global_min_index, path_t *path)
{
    uint32_t next = blockIdx.x * blockDim.x + threadIdx.x; // global ID
    int source = global_min_index[0];
    bool vertex_is_in_visibilite = false;
    double current_cost;

    // data_t edge = graph[source*VERTICES + next];
    edge_t *edge = NULL;

    for (uint32_t i = 0; i < graph[source]->nb_edges_out; i++)
    {
        if (graph[source]->out_edges[i]->id == next)
        {
            edge = graph[source]->out_edges[i];
        }
    }
    if (edge == NULL)
    {
        return;
    }

    for (uint32_t i = 0; i < path->nb_visibilite; i++)
    {
        if (path->visibilite[i] == next)
        {
            vertex_is_in_visibilite = true;
        }
    }
    if (!vertex_is_in_visibilite)
    {
        return;
    }

    current_cost = ((path->profil) * (edge->dist)) + ((1 - (path->profil)) * (edge->danger));

    if (!marked_vertex[next] &&
        (current_cost < dist_array[next]))
    {
        dist_array[next] = current_cost;
        parents[next] = source;
    }
}