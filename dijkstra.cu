#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
extern "C"{
#include "./header/edge.h"
#include "./header/path.h"
#include "./header/dijkstra.h"
#include "./header/config.h"
}

#define NUM_BLOCK  512  // Number of thread blocks
#define NUM_THREAD  512  // Number of threads per block


__global__ void dijkstra(vertex_cuda_t *graph, double *cost, int *parent, int *visited, int source, uint32_t num_vertices) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num_vertices) {
        cost[tid] = (tid == source) ? 0 : DBL_MAX;
        parent[tid] = -1;
        visited[tid] = 0;
    }
    
    __syncthreads();
    
    for (int count = 0; count < num_vertices - 1; ++count) {
        double min = DBL_MAX;
        int u = -1;
        
        for (int v = 0; v < num_vertices; ++v) {
            if (!visited[v] && cost[v] < min) {
                min = cost[v];
                u = v;
            }
        }
        
        if (u == -1) {
            break; // No reachable vertices left
        }
        
        visited[u] = 1;
        
        for (int v = 0; v < num_vertices; ++v) {
            double alt = cost[u] + graph[u * num_vertices + v].distance;
            if (!visited[v] && graph[u * num_vertices + v].distance && alt < cost[v]) {
                cost[v] = alt;
                parent[v] = u;
            }
        }
        
        __syncthreads();
    }
}

int main(int argc, char const *argv[]) {
    vertex_cuda_t *graph;
    cifre_conf_t conf;
    set_config(argv[1], &conf);
    uint32_t num_vertices;
    get_graph_cuda(&conf,&graph,&num_vertices);
    // printf("ni %d nj %d dist %f\n",1,1,graph[(1 * (num_vertices)) +1].distance);
    // printf("salut, %d\n",num_vertices);
    // for (int i = 1; i < num_vertices; i++)
    // {
    //     for (int j = 1; j < num_vertices; j++)
    //     {
    //         if(i!=j){
    //             printf("%d %d |  %2f\n",i,j,graph[i*num_vertices + j].distance);
    //         }
    //     }
        
    // }
    
    // Example graph represented as an adjacency matrix
    // int graph[num_vertices][num_vertices] = {
    //     {0, 2, 0, 6, 0},
    //     {0, 0, 3, 8, 5},
    //     {0, 0, 0, 0, 7},
    //     {0, 0, 0, 0, 9},
    //     {0, 0, 0, 0, 0}
    // };

    vertex_cuda_t *d_graph;
    double *d_cost;
    int *d_parent, *d_visited;
    double *cost = (double *) calloc(num_vertices,sizeof(double));
    int *parent = (int * )calloc(num_vertices,sizeof(int));
    int* visited = (int * )calloc(num_vertices,sizeof(int));

    // Allocate memory on GPU
    hipMalloc((void**)&d_graph, num_vertices * num_vertices * sizeof(vertex_cuda_t));
    hipMalloc((void**)&d_cost, num_vertices * sizeof(double));
    hipMalloc((void**)&d_parent, num_vertices * sizeof(int));
    hipMalloc((void**)&d_visited, num_vertices * sizeof(int));

    // Copy graph to GPU
    hipMemcpy(d_graph, graph, num_vertices * num_vertices * sizeof(vertex_cuda_t), hipMemcpyHostToDevice);

    // Choose source vertex
    int source = 1;

    // Run Dijkstra's algorithm on GPU
    dijkstra<<<1, num_vertices>>>(d_graph, d_cost, d_parent, d_visited, source,num_vertices);

    // Copy results back to CPU
    hipMemcpy(cost, d_cost, num_vertices * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(parent, d_parent, num_vertices * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(visited, d_visited, num_vertices * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    double value;
    printf("Vertex\tCost\tParent\n");
    // for (int i = 0; i < num_vertices; ++i) {
    //     if( cost[i] != DBL_MAX){
    //         printf("%d\t%f\t%d\n", i, cost[i], parent[i]);
    //     }
        
    // }
    int current = 4;
    int old_current = parent[current];
    while(current!=-1){
        printf("%d -> %d : %02f\n",current,old_current,graph[current*num_vertices+old_current].distance);
        old_current = current;
        current = parent[current];
    }
    vertex_t **graph_dij;
    edge_t **edge_array;
    uint32_t nb_vertices, nb_edges, nb_paths;
    get_graph(&conf, &graph_dij, &edge_array, &nb_vertices, &nb_edges);
    double *dist_array_ref = (double *) calloc(nb_vertices,sizeof(double));
    path_t path;
    path.destination = 4;
    path.origin = 1;
    path.profil = 1;

    double dij_cost = dijkstra_backward_full_graph(graph_dij, nb_vertices, &dist_array_ref,NULL, &path);
    printf("%f %f",dij_cost,cost[4]);
    // Free GPU memory
    hipFree(d_graph);
    hipFree(d_cost);
    hipFree(d_parent);
    hipFree(d_visited);
    free_config(&conf);
    return 0;
}
